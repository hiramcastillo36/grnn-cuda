#include <hip/hip_runtime.h>

#include <stdio.h>
#include <math.h>

struct GRNNParams {
    float sigma;
    int n_train;
    int n_features;
};

__global__ void patternLayerKernel(
    const float* X_train,
    const float* y_train,
    const float* X_test,
    float* numerator,
    float* denominator,
    const GRNNParams params,
    const int n_test
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n_test) {
        float num = 0.0f;
        float den = 0.0f;

        for(int i = 0; i < params.n_train; i++) {
            float dist = 0.0f;

            for(int j = 0; j < params.n_features; j++) {
                float diff = X_test[tid * params.n_features + j] -
                            X_train[i * params.n_features + j];
                dist += diff * diff;
            }

            float weight = exp(-dist / (2.0f * params.sigma * params.sigma));

            num += weight * y_train[i];
            den += weight;
        }

        numerator[tid] = num;
        denominator[tid] = den;
    }
}

__global__ void summationLayerKernel(
    float* numerator,
    float* denominator,
    float* output,
    const int n_test
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n_test) {
        if (denominator[tid] > 1e-10f) {
            output[tid] = numerator[tid] / denominator[tid];
        } else {
            output[tid] = 0.0f;
        }
    }
}


void predictGRNN(
    const float* X_train,
    const float* y_train,
    const float* X_test,
    float* output,
    const GRNNParams params,
    const int n_test
) {

    float *d_X_train, *d_y_train, *d_X_test;
    float *d_numerator, *d_denominator, *d_output;

    hipMalloc(&d_X_train, params.n_train * params.n_features * sizeof(float));
    hipMalloc(&d_y_train, params.n_train * sizeof(float));
    hipMalloc(&d_X_test, n_test * params.n_features * sizeof(float));
    hipMalloc(&d_numerator, n_test * sizeof(float));
    hipMalloc(&d_denominator, n_test * sizeof(float));
    hipMalloc(&d_output, n_test * sizeof(float));

    hipMemcpy(d_X_train, X_train, params.n_train * params.n_features * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y_train, y_train, params.n_train * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_X_test, X_test, n_test * params.n_features * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n_test + blockSize - 1) / blockSize;

    patternLayerKernel<<<numBlocks, blockSize>>>(
        d_X_train, d_y_train, d_X_test,
        d_numerator, d_denominator, params, n_test
    );

    summationLayerKernel<<<numBlocks, blockSize>>>(
        d_numerator, d_denominator, d_output, n_test
    );

    hipMemcpy(output, d_output, n_test * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_X_train);
    hipFree(d_y_train);
    hipFree(d_X_test);
    hipFree(d_numerator);
    hipFree(d_denominator);
    hipFree(d_output);
}
